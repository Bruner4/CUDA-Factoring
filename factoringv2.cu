#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h>
#define SIZE 512

__global__ void trial(long int *prime, long int *number, long int *length) {
	
	int i = threadIdx.x;

	if ((i > 1) && (i < *length))
	{
		if (prime[i])
		{
			long int val = i;
			if (val * val == *number)
			{
				printf("\nI divisori primi sono %ld e %ld\n", val, val);
			}
			if (*number % val == 0)
			{
				printf("\nI divisori primi sono %ld ", val);
				long int val2 = *number / val;
				printf("e %ld.\n", val2);
			}
		}
	}

}


void main()
{
	clock_t start, end;
	double tempo;
	start = clock();

	long int *d_length;
	long int *prime;
	long int *d_prime;
	int n = 2;
	long int elim;
	long int number;
	long int *d_number;

	printf("Inserisci numero da fattorizzare: ");
	scanf("%d", &number);

	long int length = floor(sqrt(number));

	prime = (long int *)malloc(SIZE * sizeof(long int));
	hipMalloc((void**) &d_prime, SIZE * sizeof(long int));
	hipMalloc((void**) &d_number, sizeof(long int));
	hipMalloc((void**) &d_length, sizeof(long int));


	for (int i = 0; i < length; i++)
		prime[i] = 1;

	while (n <= length)
	{
		if (prime[n] == 1)
		{
			elim = n + n;
			while (elim <= length)
			{
				prime[elim] = 0;
				elim += n;
			}
		}
		n++;
	}
/*	for (int i = 2; i < length; i++)
		printf("%d: %d - ", i, prime[i]);
	printf("\nLa lunghezza e' di %d\n", length);*/

	hipMemcpy(d_prime, &prime, SIZE * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_number, &number, sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_length, &length, sizeof(long int), hipMemcpyHostToDevice);

	trial << <1, 1 >> > (d_prime, d_number, d_length);

	free(prime);

	hipFree(d_prime);
	hipFree(d_number);
	hipFree(d_length);

	end = clock();
	tempo = ((double)(end - start)) / CLOCKS_PER_SEC;
	printf("Tempo: %f\n", tempo);
}
