#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <time.h>
#define SIZE 1024

__global__ void trial(long int *prime, long int *number, long int *length) {
	
	long int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i > 1)
	{
		if (prime[i])
		{
			long int val = i;
			if (val * val == *number)
			{
				printf("\nPrime factors are %ld and %ld\n", val, val);
				return;
			}
			if (*number % val == 0)
			{
				printf("\nPrime factors are %ld ", val);
				long int val2 = *number / val;
				printf("and %ld.\n", val2);
			}
		}
	}

}


void main()
{
	long int *d_length;
	long int *prime;
	long int *d_prime;
	long int n = 2;
	long int elim;
	long int number;
	long int *d_number;

	printf("Enter number to factorize: ");
	scanf("%d", &number);

	clock_t start, end;
	double tempo;
	start = clock();

	long int length = floor(sqrt(number));

	prime = (long int *)malloc(SIZE * sizeof(long int));
	hipMalloc((void**) &d_prime, SIZE * sizeof(long int));
	hipMalloc((void**) &d_number, sizeof(long int));
	hipMalloc((void**) &d_length, sizeof(long int));


	for (int i = 0; i < length; i++)
		prime[i] = 1;

	while (n <= length)
	{
		if (prime[n] == 1)
		{
			elim = n + n;
			while (elim <= length)
			{
				prime[elim] = 0;
				elim += n;
			}
		}
		n++;
	}

	hipMemcpy(d_prime, prime, SIZE * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_number, &number, sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_length, &length, sizeof(long int), hipMemcpyHostToDevice);

	trial << <1, length+1 >> > (d_prime, d_number, d_length);

	free(prime);

	hipFree(d_prime);
	hipFree(d_number);
	hipFree(d_length);

	end = clock();
	tempo = ((double)(end - start)) / CLOCKS_PER_SEC;
	printf("Tempo: %f\n", tempo);
}
